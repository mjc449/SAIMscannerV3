#include "hip/hip_runtime.h"
/**/////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                                                                            //
//  Copyright(c) 2018, Marshall Colville mjc449@cornell.edu                   //
//  All rights reserved.                                                      //
//                                                                            //
//  Redistribution and use in source and binary forms, with or without        //
//  modification, are permitted provided that the following conditions are    //
//  met :                                                                     //
//                                                                            //
//  1. Redistributions of source code must retain the above copyright notice, //
//  this list of conditions and the following disclaimer.                     //
//  2. Redistributions in binary form must reproduce the above copyright      //
//  notice, this list of conditions and the following disclaimer in the       //
//  documentation and/or other materials provided with the distribution.      //
//                                                                            //
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS       //
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED //
//  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A           //
//  PARTICULAR PURPOSE ARE DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT OWNER   //
//  OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,  //
//  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,       //
//  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR        //
//  PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF    //
//  LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING      //
//  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS        //
//  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.              //
//                                                                            //
//  The views and conclusions contained in the software and documentation are //
//  those of the authors and should not be interpreted as representing        //
//  official policies, either expressed or implied, of the SAIMScannerV3      //
//  project, the Paszek Research Group, or Cornell University.                //
//////////////////////////////////////////////////////////////////////////////*/

#include "saim_model_gpu.h"
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#define __CUDA_INTERNAL_COMPILATION__
//#include <common_functions.h>
//#include <math_functions.h>
#include <hip/hip_math_constants.h>
#undef __CUDA_INTERNAL_COMPILATION__
#include <complex>

namespace saim_model_gpu
{
#define checkCuda(ans){cudaAssert((ans), __FILE__, __LINE__);}
   inline void cudaAssert(
      hipError_t code,
      const char *file,
      int line,
      bool abort = true)
   {
      if (code != hipSuccess)
      {
         fprintf(stderr, "GPUassert: %s, %s, %d\n", hipGetErrorString(code), file, line);
         if (abort)
            exit(code);
      }
   }

   __global__ void EvaluateFunction(int emptyThreads, int samples, unsigned short *data, double *xvec, double *fvec, double *constants)
   {
      if ((blockIdx.x + 1 == gridDim.x) && (threadIdx.x > blockDim.x - emptyThreads))
         return;
      int tid = threadIdx.x + blockDim.x * blockIdx.x;
      int frame = (tid % samples);
      int pixel = tid / samples;
      double A{ xvec[pixel] }, B{ xvec[pixel + 1] }, H{ xvec[pixel + 2] };
      double c{ constants[frame * 3] }, d{ constants[frame * 3 + 1] }, phi{ constants[frame * 3 + 2] };
      double fval = A * (1 + 2 * c * cos(phi * H) - 2 * d * sin(phi * H) + c * c + d * d) + B;
      fvec[tid] = fval - data[tid];
   }

   __global__ void EvaluateJacobian(int emptyThreads, int samples, double *xvec, double *jvec, double *constants)
   {
      if ((blockIdx.x + 1 == gridDim.x) && (threadIdx.x > blockDim.x - emptyThreads))
         return;
      int tid = threadIdx.x + blockDim.x * blockIdx.x;
      int frame = (tid % samples);
      int pixel = tid / samples;
      int totalPixels = blockDim.x * gridDim.x / samples;
      int jidx = (pixel * 3) + (totalPixels * 3) * (pixel * samples + frame);
      double A{ xvec[pixel] }, H{ xvec[pixel + 2] };
      double c{ constants[frame * 3] }, d{ constants[frame * 3 + 1] }, phi{ constants[frame * 3 + 2] };
      jvec[jidx] = 1 + 2 * c * cos(phi * H) - 2 * d * sin(phi * H) + c * c + d * d;
      jvec[jidx + 1] = 1;
      jvec[jidx + 2] = -2 * A * phi * (c * sin(phi * H) + d * cos(phi * H));
   }

   GPUModel::GPUModel()
   {
      _initialized = false;
   }

   GPUModel::~GPUModel()
   {

   }

   int GPUModel::RegisterImages(std::vector<cv::Mat> &input)
   {
      _rawImgs = input;
      for (int i = 0; i < 3; i++)
      {
         _outputImgs.push_back(cv::Mat(_rawImgs[0].rows, _rawImgs[0].cols, CV_64F));
      }
      return 0;
   }

   int GPUModel::SetGrainSize(int val)
   {
      if (_initialized)
         return 1;
      _grainSize = val;
      return 0;
   }

   int GPUModel::InitializeBuffers()
   {
      //Setup the device - assumes that the system has 2 devices and uses the second
      checkCuda(hipSetDevice(1));
      checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync | hipDeviceMapHost));

      _m = _rawImgs[0].rows * _rawImgs[0].cols;
      _n = _rawImgs.size();
      //Buffer sizes
      _datasz = _m * _n;
      _ngrains = _datasz % _grainSize == 0 ? _datasz / _grainSize : _datasz / _grainSize + 1;
      _xsz = _grainSize * 3;
      _fnsz = _grainSize * _n;
      _jacsz = _fnsz * _grainSize * 3;
      //Host side allocations in pinned memory
      checkCuda(hipHostAlloc((void **)&_h_data, _datasz * sizeof(unsigned short), hipHostMallocMapped | hipHostMallocWriteCombined));
      checkCuda(hipHostAlloc((void **)&_h_xvec, _xsz * sizeof(double), hipHostMallocMapped | hipHostMallocWriteCombined));
      checkCuda(hipHostAlloc((void **)&_h_fvec, _fnsz * sizeof(double), hipHostMallocMapped));
      checkCuda(hipHostAlloc((void **)&_h_jvec, _jacsz * sizeof(double), hipHostMallocMapped));
      checkCuda(hipHostAlloc((void **)&_h_constvec, 3 * _n * sizeof(double), hipHostMallocMapped | hipHostMallocWriteCombined));
      //Device side allocations
      checkCuda(hipHostGetDevicePointer(&_d_data, _h_data, 0));
      checkCuda(hipHostGetDevicePointer(&_d_xvec, _h_xvec, 0));
      checkCuda(hipHostGetDevicePointer(&_d_fvec, _h_fvec, 0));
      checkCuda(hipHostGetDevicePointer(&_d_jvec, _h_jvec, 0));
      checkCuda(hipHostGetDevicePointer(&_d_constvec, _h_constvec, 0));

      for (int i = 0; i < _jacsz; i++)
      {
         _h_jvec[i] = 0.0;
      }

      //Calculate the number of leftover threads for the FN and Jac kernel launches
      if (_fnsz % 512 == 0)
      {
         _blocksPerGrid = _fnsz / 512;
         _emptyThreads = 0;
      }
      else
      {
         _blocksPerGrid = _fnsz / 512 + 1;
         _emptyThreads = 512 - (_fnsz % 512);
      }

      if (_datasz % _grainSize == 0)
      {
         _ngrains = _datasz / _grainSize;
         _emptyPixels = 0;
      }
      else
      {
         _ngrains = _datasz / _grainSize + 1;
         _emptyPixels = _grainSize - _datasz % _grainSize;
      }
      _nVars = _grainSize * 3;
      _mPoints = _fnsz;
      
      for (int i = 0; i < _rawImgs.size(); i++)
      {
         unsigned short *ptr = _rawImgs[i].ptr<unsigned short>();
         for (int j = 0; j < _m; j++)
         {
            _h_data[j * _n + i] = ptr[j];
         }
      }

      _initialized = true;
      return 0;
   }

   int GPUModel::ReleaseBuffers(void)
   {
      checkCuda(hipHostFree(_h_data));
      checkCuda(hipHostFree(_h_xvec));
      checkCuda(hipHostFree(_h_fvec));
      checkCuda(hipHostFree(_h_jvec));
      checkCuda(hipHostFree(_h_constvec));
      _initialized = false;
      return 0;
   }

   int GPUModel::CalculateConstants(double wavelength, double dOx, double nB, double nOx, double nSi, double *angles)
   {
      double angleOx, angleSi, pB, pOx, pSi, kOx, m11, m12, m21, m22;
      std::complex<double> num, denom, rTE;
      for (int i = 0; i < _n; i++)
      {
         angleOx = asin(sin(angles[i]) * nB / nOx);
         angleSi = asin(sin(angleOx) * nOx / nSi);
         kOx = 2.0 * HIP_PI * nOx / wavelength;
         pB = nB * cos(angles[i]);
         pOx = nOx * cos(angleOx);
         pSi = nSi * cos(angleSi);
         m11 = cos(kOx * dOx * cos(angleOx));
         m12 = -1.0 / pOx * sin(kOx * dOx * cos(angleOx));
         m21 = -pOx * sin(kOx * dOx * cos(angleOx));
         m22 = m11;
         num = std::complex<double>(m11 * pB - m22 * pSi, m12 * pSi * pB + m21);
         denom = std::complex<double>(m11 * pB + m22 * pSi, m12 * pSi * pB + m21);
         rTE = num / denom;
         _h_constvec[i * 3] = rTE.real();
         _h_constvec[i * 3 + 1] = rTE.imag();
         _h_constvec[i * 3 + 2] = 4 * HIP_PI * nB * cos(angles[i]) / wavelength;
      }
      return 0;
   }

   int GPUModel::CalculateFunction(int grain, int emptyThreads)
   {
      hipEvent_t stop, start;
      checkCuda(hipEventCreateWithFlags(&start, hipEventBlockingSync));
      checkCuda(hipEventCreateWithFlags(&stop, hipEventBlockingSync));
      checkCuda(hipEventRecord(start));

      unsigned short *dataptr = _d_data + grain * _grainSize;

      EvaluateFunction<<<_blocksPerGrid, 512>>>(emptyThreads, _n, dataptr, _d_xvec, _d_fvec, _d_constvec);

      checkCuda(hipEventRecord(stop));
      checkCuda(hipEventSynchronize(stop));
      float milliseconds;
      checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
      //std::cout << "Function calculation took: " << milliseconds << " milliseconds.\n";

      return 0;
   }

   int GPUModel::CalculateJacobian(int grain, int emptyThreads)
   {
      hipEvent_t stop, start;
      checkCuda(hipEventCreateWithFlags(&stop, hipEventBlockingSync));
      checkCuda(hipEventCreateWithFlags(&start, hipEventBlockingSync));
      checkCuda(hipEventRecord(start));

      EvaluateJacobian<<<_blocksPerGrid, 512>>>(emptyThreads, _n, _d_xvec, _d_jvec, _d_constvec);

      checkCuda(hipEventRecord(stop));
      checkCuda(hipEventSynchronize(stop));
      float milliseconds;
      checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
      //std::cout << "Jacobian calculation took: " << milliseconds << " milliseconds.\n";

      return 0;
   }

   int GPUModel::RunFit(void)
   {
      std::chrono::high_resolution_clock::time_point earlier, later;
      std::chrono::duration<double> timeTaken;
      //Run the solver on each grain
      for (int i = 0; i < _ngrains; i++)
      {
         earlier = std::chrono::high_resolution_clock::now();
         for (int i = 0; i < _grainSize; i++)
         {
            _h_xvec[i * 3] = 200;
            _h_xvec[i * 3 + 1] = 100;
            _h_xvec[i * 3 + 2] = 6.5;
         }

         if (dtrnlsp_init(&_solverHandle, &_nVars, &_mPoints, _h_xvec, _eps, &_iterations, &_stepIterations, &_initialStep) != TR_SUCCESS)
         {
            std::cerr << "Error initializing solver" << std::endl;
            MKL_Free_Buffers();
            return 1;
         }
         if (dtrnlsp_check(&_solverHandle, &_nVars, &_mPoints, _h_jvec, _h_fvec, _eps, _fitInfo) != TR_SUCCESS)
         {
            std::cerr << "Error checking solver" << std::endl;
            MKL_Free_Buffers();
            return 2;
         }
         else
         {
            if (_fitInfo[0] != 0 ||
               _fitInfo[1] != 0 ||
               _fitInfo[2] != 0 ||
               _fitInfo[3] != 0)
            {
               std::cerr << "Invalid array passed to solver: " << std::endl;
               MKL_Free_Buffers();
               return 3;
            }
         }
         _successful = 0;
         _counter = 0;
         while (_successful == 0)
         {
            int emptyThreads;
            if (i == _ngrains - 1)
               emptyThreads = _emptyThreads;
            else
               emptyThreads = 0;
            if (dtrnlsp_solve(&_solverHandle, _h_fvec, _h_jvec, &_rciRequest) != TR_SUCCESS)
            {
               std::cerr << "Error solving solver" << std::endl;
               MKL_Free_Buffers();
               return 3;
            }
            if (_rciRequest == -1 ||
               _rciRequest == -2 ||
               _rciRequest == -3 ||
               _rciRequest == -4 ||
               _rciRequest == -5 ||
               _rciRequest == -6)
               _successful = 1;
            if (_rciRequest == 1)
               CalculateFunction(i, emptyThreads);
            if (_rciRequest == 2)
               CalculateJacobian(i, emptyThreads);
            //std::cout << "RCI cycle: " << _counter++ << std::endl;
         }
         if (dtrnlsp_get(&_solverHandle, &_actualIterations, &_stopCrit, &_initialRes, &_finalRes) != TR_SUCCESS)
         {
            std::cerr << "Error getting solver results" << std::endl;
            MKL_Free_Buffers();
            return 4;
         }
         if (dtrnlsp_delete(&_solverHandle) != TR_SUCCESS)
         {
            std::cerr << "Error deleting the solver" << std::endl;
            MKL_Free_Buffers();
            return 5;
         }

         MKL_Free_Buffers();
         double *aptr, *bptr, *hptr;
         aptr = _outputImgs[0].ptr<double>() + i * _grainSize;
         bptr = _outputImgs[1].ptr<double>() + i * _grainSize;
         hptr = _outputImgs[2].ptr<double>() + i * _grainSize;
         for (int j = 0; j < _grainSize; j++)
         {
            int xidx = j * 3;
            aptr[j] = _h_xvec[xidx];
            bptr[j] = _h_xvec[xidx + 1];
            hptr[j] = _h_xvec[xidx + 2];
         }
         later = std::chrono::high_resolution_clock::now();
         timeTaken = later - earlier;
         std::cout << "Grain " << i << " of " << _ngrains << " finished in " << std::chrono::duration_cast<std::chrono::milliseconds>(timeTaken).count() <<" milliseconds." << std::endl;
      }

      return 0;
   }
}